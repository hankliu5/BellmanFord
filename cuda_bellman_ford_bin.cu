/*
 * This is a CUDA version of bellman_ford algorithm
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_bellman_ford cuda_bellman_ford.cu
 * Run: ./cuda_bellman_ford <input file> <number of blocks per grid> <number of threads per block>, you will find the output file 'output.txt'
 * */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <sys/time.h>
#include <climits>

#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

// for mmap
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

// for timing
#include <sys/time.h>

using std::string;
using std::cout;
using std::endl;

#define INF INT_MAX
#define THREADS_PER_BLOCK 1024

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
{                                                                              \
	const hipError_t error = call;                                            \
	if (error != hipSuccess)                                                  \
	{                                                                          \
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
		fprintf(stderr, "code: %d, reason: %s\n", error,                       \
				hipGetErrorString(error));                                    \
				exit(1);                                                       \
	}                                                                          \
}

void get_outedges(int64_t* graph, int64_t* outedges, size_t interval_st, size_t num_of_subvertices, size_t num_of_vertices) {
    memcpy(outedges, (graph + interval_st * num_of_vertices), sizeof(int64_t) * num_of_subvertices * num_of_vertices);
}

void get_inedges(int64_t* graph, int64_t* inedges, size_t interval_st, size_t interval_en, size_t num_of_vertices) {
    size_t i, j, row;
    int64_t *graph_ptr, *inedges_ptr;
    // in column favor but transpose into row.
    for (i = 0; i < num_of_vertices; i++) {
        for (row = 0, j = interval_st; j < interval_en; row++, j++) {
            // printf("i: %lu, j: %lu, row: %lu\n", i, j, row);
            *(inedges + row * num_of_vertices + i) = *(graph + i * num_of_vertices + j);
        }
    }
}

__global__ void bellman_ford_one_iter(size_t n, int64_t *d_mat, int64_t *d_dist, bool *d_has_next){
	size_t global_tid = blockDim.x * blockIdx.x + threadIdx.x;
	size_t v = global_tid;
	size_t u;
	if (global_tid >= n) return;
	for(u = 0; u < n; u++){
		int64_t weight = d_mat[u * n + v]; // row is src, col is dst
		if (weight > 0) {
			int64_t new_dist = d_dist[u] + weight;
			if(new_dist < d_dist[v]){
				d_dist[v] = new_dist;
				*d_has_next = true;
			}
		}
	}
}

/**
 * Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
 * @param blockPerGrid number of blocks per grid
 * @param threadsPerBlock number of threads per block
 * @param n input size
 * @param *mat input adjacency matrix
 * @param *dist distance array
 * @param *has_negative_cycle a bool variable to recode if there are negative cycles
 */
void bellman_ford(size_t n, int64_t *mat, int64_t *dist, bool *has_negative_cycle) {
	size_t iter_num = 0;
	int64_t *d_mat, *d_dist;
	bool *d_has_next, h_has_next;
	size_t i;

	hipMalloc(&d_mat, sizeof(int64_t) * n * n);
	hipMalloc(&d_dist, sizeof(int64_t) * n);
	hipMalloc(&d_has_next, sizeof(bool));

	*has_negative_cycle = false;

	for(i = 0 ; i < n; i++){
		dist[i] = INF;
	}

	dist[0] = 0;
	hipMemcpy(d_mat, mat, sizeof(int64_t) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(d_dist, dist, sizeof(int64_t) * n, hipMemcpyHostToDevice);

	do {
		h_has_next = false;
		hipMemcpy(d_has_next, &h_has_next, sizeof(bool), hipMemcpyHostToDevice);

		bellman_ford_one_iter<<<(n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(n, d_mat, d_dist, d_has_next);
		CHECK(hipDeviceSynchronize());
		hipMemcpy(&h_has_next, d_has_next, sizeof(bool), hipMemcpyDeviceToHost);

		iter_num++;
		if(iter_num >= n-1){
			*has_negative_cycle = true;
			break;
		}
	} while (h_has_next);

	if(! *has_negative_cycle){
		hipMemcpy(dist, d_dist, sizeof(int64_t) * n, hipMemcpyDeviceToHost);
	}

	hipFree(d_mat);
	hipFree(d_dist);
	hipFree(d_has_next);
}

/**
 * TODO section:
 * maybe we can borrow the log system from graphchi?
 */
int main(int argc, char** argv) {
    int64_t *graph, *outedges, *inedges;
    int fd;
    size_t num_of_vertices, num_of_subvertices, niters;
    size_t iter, st, i;
    
    // result
    int64_t *vertices;
	bool has_negative_cycle = false;

    // timing
    struct timeval h_start, h_end;
    long duration;

    if (argc < 4) {
        printf("usage: %s <graph path> <# of vertices> <# of subvertices>\n", argv[0]);
        exit(1);
    } 

    // I/O part, open in mmap mode
    fd = open(argv[1], O_RDONLY);
    num_of_vertices = (size_t) atoi(argv[2]);
    num_of_subvertices = (size_t) atoi(argv[3]);
    niters = (size_t) atoi(argv[4]);
    graph = (int64_t *) mmap(NULL, sizeof(int64_t) * num_of_vertices * num_of_vertices, PROT_READ, MAP_PRIVATE, fd, 0);

    // calculate the largest stripe we can have
    // Assume we have 1 GB (like graphchi), and at least we can contain one row and one column of the graph
    // and assume we those numbers are the power of 2
    const size_t stripe_sz = num_of_vertices * num_of_subvertices * sizeof(int64_t);
    // num_of_subvertices = 32;
    printf("num_of_subvertices: %lu\n", num_of_subvertices);

    // subgraph initialization
    outedges = (int64_t *) malloc(stripe_sz);
    inedges = (int64_t *) malloc(stripe_sz);
    printf("graph: %p, outedges: %p, inedges: %p\n", graph, outedges, inedges);

    // PR initialization
    vertices = (int64_t *) calloc(sizeof(int64_t), num_of_vertices);
    
    for (i = 0; i < num_of_vertices; i++) {
        vertices[i] = INF;
    }

	bellman_ford(num_of_vertices, graph, vertices, &has_negative_cycle);

    FILE *fp = fopen("log.txt", "w");
    for (i = 0; i < num_of_vertices; i++) {
        fprintf(fp, "%lu %lu\n", i, vertices[i]);
    }
    fclose(fp);
    // cleanup
    munmap(graph, sizeof(int64_t) * num_of_vertices * num_of_vertices);
    close(fd);

    free(outedges);
    free(inedges);
    free(vertices);

    return 0;
}